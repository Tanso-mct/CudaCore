﻿#include "CudaCore/include/pch.h"
#include "CudaCore/include/memory.cuh"

#include "CudaCore/include/console.cuh"

CUDA_CORE bool CudaCore::CheckCudaErr(hipError_t call)
{
    if (call != hipSuccess) return false;
    return true;
}

namespace
{

std::string GetPointerAddress(const void *ptr)
{
    std::stringstream ss;
    ss << std::hex << std::uppercase << std::setw(16) << std::setfill('0') << reinterpret_cast<uintptr_t>(ptr);
    return ss.str();
}

}

CUDA_CORE void CudaCore::Malloc(void **devPtr, size_t size)
{
    if (CheckCudaErr(hipMalloc(devPtr, size)))
    {
#ifndef NDEBUG
        CudaCore::CoutDebug
        ({
            "CUDA malloc succeeded.", 
            "address:" + ::GetPointerAddress(*devPtr),
            "size:" + std::to_string(size),
        });
#endif
    }
    else
    {
        hipError_t err = hipGetLastError();
        CudaCore::CoutErr
        ({
            "CUDA malloc failed.", 
            "code:" + std::to_string(err),
            "reason:" + std::string(hipGetErrorString(err))
        });

        throw std::runtime_error("CudaCore::Malloc failed");
    }
}

CUDA_CORE bool CudaCore::TryMalloc(void **devPtr, size_t size)
{
    if (CheckCudaErr(hipMalloc(devPtr, size)))
    {
#ifndef NDEBUG
        CudaCore::CoutDebug
        ({
            "CUDA malloc succeeded.", 
            "address:" + ::GetPointerAddress(*devPtr),
            "size:" + std::to_string(size),
        });
#endif
        return true;
    }
    else
    {
        hipError_t err = hipGetLastError();
        CudaCore::CoutErr
        ({
            "CUDA malloc failed.", 
            "code:" + std::to_string(err),
            "reason:" + std::string(hipGetErrorString(err))
        });

        return false;
    }
}

CUDA_CORE void CudaCore::Free(void **devPtr)
{
    if (CheckCudaErr(hipFree(*devPtr)))
    {
#ifndef NDEBUG
        CudaCore::CoutDebug
        ({
            "CUDA free succeeded.", 
            "address:" + ::GetPointerAddress(*devPtr),
        });
#endif
        *devPtr = nullptr;
    }
    else
    {
        hipError_t err = hipGetLastError();
        CudaCore::CoutErr
        ({
            "CUDA free failed.", 
            "code:" + std::to_string(err),
            "reason:" + std::string(hipGetErrorString(err))
        });

        throw std::runtime_error("CudaCore::Free failed");
    }
}
CUDA_CORE bool CudaCore::TryFree(void **devPtr)
{
    if (CheckCudaErr(hipFree(*devPtr)))
    {
#ifndef NDEBUG
        CudaCore::CoutDebug
        ({
            "CUDA free succeeded.", 
            "address:" + ::GetPointerAddress(*devPtr),
        });
#endif
        *devPtr = nullptr;
        return true;
    }
    else
    {
        hipError_t err = hipGetLastError();
        CudaCore::CoutErr
        ({
            "CUDA free failed.", 
            "code:" + std::to_string(err),
            "reason:" + std::string(hipGetErrorString(err))
        });

        return false;
    }
}

CUDA_CORE void CudaCore::MallocHost(void **ptr, size_t size)
{
    if (CheckCudaErr(hipHostMalloc(ptr, size)))
    {
#ifndef NDEBUG
        CudaCore::CoutDebug
        ({
            "CUDA mallocHost succeeded.", 
            "address:" + ::GetPointerAddress(*ptr),
            "size:" + std::to_string(size),
        });
#endif
    }
    else
    {
        hipError_t err = hipGetLastError();
        CudaCore::CoutErr
        ({
            "CUDA mallocHost failed.", 
            "code:" + std::to_string(err),
            "reason:" + std::string(hipGetErrorString(err))
        });

        throw std::runtime_error("CudaCore::MallocHost failed");
    }
}

CUDA_CORE bool CudaCore::TryMallocHost(void **ptr, size_t size)
{
    if (CheckCudaErr(hipHostMalloc(ptr, size)))
    {
#ifndef NDEBUG
        CudaCore::CoutDebug
        ({
            "CUDA mallocHost succeeded.", 
            "address:" + ::GetPointerAddress(*ptr),
            "size:" + std::to_string(size),
        });
#endif
        return true;
    }
    else
    {
        hipError_t err = hipGetLastError();
        CudaCore::CoutErr
        ({
            "CUDA mallocHost failed.", 
            "code:" + std::to_string(err),
            "reason:" + std::string(hipGetErrorString(err))
        });

        return false;
    }
}

CUDA_CORE void CudaCore::FreeHost(void **ptr)
{
    if (CheckCudaErr(hipHostFree(*ptr)))
    {
#ifndef NDEBUG
        CudaCore::CoutDebug
        ({
            "CUDA freeHost succeeded.", 
            "address:" + ::GetPointerAddress(*ptr),
        });
#endif
        *ptr = nullptr;
    }
    else
    {
        hipError_t err = hipGetLastError();
        CudaCore::CoutErr
        ({
            "CUDA freeHost failed.", 
            "code:" + std::to_string(err),
            "reason:" + std::string(hipGetErrorString(err))
        });

        throw std::runtime_error("CudaCore::FreeHost failed");
    }
}

CUDA_CORE bool CudaCore::TryFreeHost(void **ptr)
{
    if (CheckCudaErr(hipHostFree(*ptr)))
    {
#ifndef NDEBUG
        CudaCore::CoutDebug
        ({
            "CUDA freeHost succeeded.", 
            "address:" + ::GetPointerAddress(*ptr),
        });
#endif
        *ptr = nullptr;
        return true;
    }
    else
    {
        hipError_t err = hipGetLastError();
        CudaCore::CoutErr
        ({
            "CUDA freeHost failed.", 
            "code:" + std::to_string(err),
            "reason:" + std::string(hipGetErrorString(err))
        });

        return false;
    }
}

CUDA_CORE void CudaCore::MallocManaged(void **devPtr, size_t size)
{
    if (CheckCudaErr(hipMallocManaged(devPtr, size)))
    {
#ifndef NDEBUG
        CudaCore::CoutDebug
        ({
            "CUDA mallocManaged succeeded.", 
            "address:" + ::GetPointerAddress(*devPtr),
            "size:" + std::to_string(size),
        });
#endif
    }
    else
    {
        hipError_t err = hipGetLastError();
        CudaCore::CoutErr
        ({
            "CUDA mallocManaged failed.", 
            "code:" + std::to_string(err),
            "reason:" + std::string(hipGetErrorString(err))
        });

        throw std::runtime_error("CudaCore::MallocManaged failed");
    }
}

CUDA_CORE bool CudaCore::TryMallocManaged(void **devPtr, size_t size)
{
    if (CheckCudaErr(hipMallocManaged(devPtr, size)))
    {
#ifndef NDEBUG
        CudaCore::CoutDebug
        ({
            "CUDA mallocManaged succeeded.", 
            "address:" + ::GetPointerAddress(*devPtr),
            "size:" + std::to_string(size),
        });
#endif
        return true;
    }
    else
    {
        hipError_t err = hipGetLastError();
        CudaCore::CoutErr
        ({
            "CUDA mallocManaged failed.", 
            "code:" + std::to_string(err),
            "reason:" + std::string(hipGetErrorString(err))
        });

        return false;
    }
}

CUDA_CORE void CudaCore::FreeManaged(void **devPtr)
{
    if (CheckCudaErr(hipFree(*devPtr)))
    {
#ifndef NDEBUG
        CudaCore::CoutDebug
        ({
            "CUDA freeManaged succeeded.", 
            "address:" + ::GetPointerAddress(*devPtr),
        });
#endif
        *devPtr = nullptr;
    }
    else
    {
        hipError_t err = hipGetLastError();
        CudaCore::CoutErr
        ({
            "CUDA freeManaged failed.", 
            "code:" + std::to_string(err),
            "reason:" + std::string(hipGetErrorString(err))
        });

        throw std::runtime_error("CudaCore::FreeManaged failed");
    }
}

CUDA_CORE bool CudaCore::TryFreeManaged(void **devPtr)
{
    if (CheckCudaErr(hipFree(*devPtr)))
    {
#ifndef NDEBUG
        CudaCore::CoutDebug
        ({
            "CUDA freeManaged succeeded.", 
            "address:" + ::GetPointerAddress(*devPtr),
        });
#endif
        *devPtr = nullptr;
        return true;
    }
    else
    {
        hipError_t err = hipGetLastError();
        CudaCore::CoutErr
        ({
            "CUDA freeManaged failed.", 
            "code:" + std::to_string(err),
            "reason:" + std::string(hipGetErrorString(err))
        });

        return false;
    }
}

CUDA_CORE void CudaCore::MallocArray
(
    hipArray_t *array, const hipChannelFormatDesc *desc, 
    size_t width, size_t height, unsigned int flags
){
    if (CheckCudaErr(hipMallocArray(array, desc, width, height, flags)))
    {
#ifndef NDEBUG
        CudaCore::CoutDebug
        ({
            "CUDA mallocArray succeeded.", 
            "address:" + ::GetPointerAddress(*array),
            "width:" + std::to_string(width),
            "height:" + std::to_string(height),
        });
#endif
    }
    else
    {
        hipError_t err = hipGetLastError();
        CudaCore::CoutErr
        ({
            "CUDA mallocArray failed.", 
            "code:" + std::to_string(err),
            "reason:" + std::string(hipGetErrorString(err))
        });

        throw std::runtime_error("CudaCore::MallocArray failed");
    }
}

CUDA_CORE bool CudaCore::TryMallocArray
(
    hipArray_t *array, const hipChannelFormatDesc *desc, 
    size_t width, size_t height, unsigned int flags
){
    if (CheckCudaErr(hipMallocArray(array, desc, width, height, flags)))
    {
#ifndef NDEBUG
        CudaCore::CoutDebug
        ({
            "CUDA mallocArray succeeded.", 
            "address:" + ::GetPointerAddress(*array),
            "width:" + std::to_string(width),
            "height:" + std::to_string(height),
        });
#endif
        return true;
    }
    else
    {
        hipError_t err = hipGetLastError();
        CudaCore::CoutErr
        ({
            "CUDA mallocArray failed.", 
            "code:" + std::to_string(err),
            "reason:" + std::string(hipGetErrorString(err))
        });

        return false;
    }
}

CUDA_CORE void CudaCore::FreeArray(hipArray_t *array)
{
    if (CheckCudaErr(hipFreeArray(*array)))
    {
#ifndef NDEBUG
        CudaCore::CoutDebug
        ({
            "CUDA freeArray succeeded.", 
            "address:" + ::GetPointerAddress(*array),
        });
#endif
        *array = nullptr;
    }
    else
    {
        hipError_t err = hipGetLastError();
        CudaCore::CoutErr
        ({
            "CUDA freeArray failed.", 
            "code:" + std::to_string(err),
            "reason:" + std::string(hipGetErrorString(err))
        });

        throw std::runtime_error("CudaCore::FreeArray failed");
    }
}

CUDA_CORE bool CudaCore::TryFreeArray(hipArray_t *array)
{
    if (CheckCudaErr(hipFreeArray(*array)))
    {
#ifndef NDEBUG
        CudaCore::CoutDebug
        ({
            "CUDA freeArray succeeded.", 
            "address:" + ::GetPointerAddress(*array),
        });
#endif
        *array = nullptr;
        return true;
    }
    else
    {
        hipError_t err = hipGetLastError();
        CudaCore::CoutErr
        ({
            "CUDA freeArray failed.", 
            "code:" + std::to_string(err),
            "reason:" + std::string(hipGetErrorString(err))
        });

        return false;
    }
}

CUDA_CORE void CudaCore::Memcpy(void *dst, const void *src, size_t size, hipMemcpyKind kind)
{
    if (CheckCudaErr(hipMemcpy(dst, src, size, kind)))
    {
#ifndef NDEBUG
        CudaCore::CoutDebug
        ({
            "CUDA memcpy succeeded.", 
            "dst address:" + ::GetPointerAddress(dst),
            "src address:" + ::GetPointerAddress(src),
            "size:" + std::to_string(size),
        });
#endif
    }
    else
    {
        hipError_t err = hipGetLastError();
        CudaCore::CoutErr
        ({
            "CUDA memcpy failed.", 
            "code:" + std::to_string(err),
            "reason:" + std::string(hipGetErrorString(err))
        });

        throw std::runtime_error("CudaCore::Memcpy failed");
    }
}

CUDA_CORE bool CudaCore::TryMemcpy(void *dst, const void *src, size_t size, hipMemcpyKind kind)
{
    if (CheckCudaErr(hipMemcpy(dst, src, size, kind)))
    {
#ifndef NDEBUG
        CudaCore::CoutDebug
        ({
            "CUDA memcpy succeeded.", 
            "dst address:" + ::GetPointerAddress(dst),
            "src address:" + ::GetPointerAddress(src),
            "size:" + std::to_string(size),
        });
#endif
        return true;
    }
    else
    {
        hipError_t err = hipGetLastError();
        CudaCore::CoutErr
        ({
            "CUDA memcpy failed.", 
            "code:" + std::to_string(err),
            "reason:" + std::string(hipGetErrorString(err))
        });

        return false;
    }
}

CUDA_CORE void CudaCore::Memset(void *devPtr, int value, size_t size)
{
    if (CheckCudaErr(hipMemset(devPtr, value, size)))
    {
#ifndef NDEBUG
        CudaCore::CoutDebug
        ({
            "CUDA memset succeeded.", 
            "address:" + ::GetPointerAddress(devPtr),
            "value:" + std::to_string(value),
            "size:" + std::to_string(size),
        });
#endif
    }
    else
    {
        hipError_t err = hipGetLastError();
        CudaCore::CoutErr
        ({
            "CUDA memset failed.", 
            "code:" + std::to_string(err),
            "reason:" + std::string(hipGetErrorString(err))
        });

        throw std::runtime_error("CudaCore::Memset failed");
    }
}

CUDA_CORE bool CudaCore::TryMemset(void *devPtr, int value, size_t size)
{
    if (CheckCudaErr(hipMemset(devPtr, value, size)))
    {
#ifndef NDEBUG
        CudaCore::CoutDebug
        ({
            "CUDA memset succeeded.", 
            "address:" + ::GetPointerAddress(devPtr),
            "value:" + std::to_string(value),
            "size:" + std::to_string(size),
        });
#endif
        return true;
    }
    else
    {
        hipError_t err = hipGetLastError();
        CudaCore::CoutErr
        ({
            "CUDA memset failed.", 
            "code:" + std::to_string(err),
            "reason:" + std::string(hipGetErrorString(err))
        });

        return false;
    }
}

CUDA_CORE void CudaCore::CreateSurfaceObj(hipSurfaceObject_t *obj, const hipResourceDesc *rResDesc)
{
    if (CheckCudaErr(hipCreateSurfaceObject(obj, rResDesc)))
    {
#ifndef NDEBUG
        CudaCore::CoutDebug
        ({
            "CUDA create surface object succeeded.", 
            "object:" + std::to_string(*obj),
            "array address:" + ::GetPointerAddress(rResDesc->res.array.array),
        });
#endif
    }
    else
    {
        hipError_t err = hipGetLastError();
        CudaCore::CoutErr
        ({
            "CUDA create surface object failed.", 
            "code:" + std::to_string(err),
            "reason:" + std::string(hipGetErrorString(err))
        });

        throw std::runtime_error("CudaCore::CreateSurfaceObj failed");
    }
}

CUDA_CORE bool CudaCore::TryCreateSurfaceObj(hipSurfaceObject_t *obj, const hipResourceDesc *rResDesc)
{
    if (CheckCudaErr(hipCreateSurfaceObject(obj, rResDesc)))
    {
#ifndef NDEBUG
        CudaCore::CoutDebug
        ({
            "CUDA create surface object succeeded.", 
            "object:" + std::to_string(*obj),
            "array address:" + ::GetPointerAddress(rResDesc->res.array.array),
        });
#endif
        return true;
    }
    else
    {
        hipError_t err = hipGetLastError();
        CudaCore::CoutErr
        ({
            "CUDA create surface object failed.", 
            "code:" + std::to_string(err),
            "reason:" + std::string(hipGetErrorString(err))
        });

        return false;
    }
}

CUDA_CORE void CudaCore::DestroySurfaceObj(hipSurfaceObject_t *obj)
{
    if (CheckCudaErr(hipDestroySurfaceObject(*obj)))
    {
#ifndef NDEBUG
        CudaCore::CoutDebug
        ({
            "CUDA destroy surface object succeeded.", 
            "object:" + std::to_string(*obj),
        });
#endif
        *obj = 0;
    }
    else
    {
        hipError_t err = hipGetLastError();
        CudaCore::CoutErr
        ({
            "CUDA destroy surface object failed.", 
            "code:" + std::to_string(err),
            "reason:" + std::string(hipGetErrorString(err))
        });

        throw std::runtime_error("CudaCore::DestroySurfaceObj failed");
    }
}

CUDA_CORE bool CudaCore::TryDestroySurfaceObj(hipSurfaceObject_t *obj)
{
    if (CheckCudaErr(hipDestroySurfaceObject(*obj)))
    {
#ifndef NDEBUG
        CudaCore::CoutDebug
        ({
            "CUDA destroy surface object succeeded.", 
            "object:" + std::to_string(*obj),
        });
#endif
        *obj = 0;
        return true;
    }
    else
    {
        hipError_t err = hipGetLastError();
        CudaCore::CoutErr
        ({
            "CUDA destroy surface object failed.", 
            "code:" + std::to_string(err),
            "reason:" + std::string(hipGetErrorString(err))
        });

        return false;
    }
}

CUDA_CORE void CudaCore::CreateTextureObj
(
    hipTextureObject_t *obj, 
    const hipResourceDesc *resDesc, 
    const hipTextureDesc *texDesc, 
    const hipResourceViewDesc *resViewDesc
){
    if (CheckCudaErr(hipCreateTextureObject(obj, resDesc, texDesc, resViewDesc)))
    {
#ifndef NDEBUG
        CudaCore::CoutDebug
        ({
            "CUDA create texture object succeeded.", 
            "object:" + std::to_string(*obj),
            "array address:" + ::GetPointerAddress(resDesc->res.array.array),
        });
#endif
    }
    else
    {
        hipError_t err = hipGetLastError();
        CudaCore::CoutErr
        ({
            "CUDA create texture object failed.", 
            "code:" + std::to_string(err),
            "reason:" + std::string(hipGetErrorString(err))
        });

        throw std::runtime_error("CudaCore::CreateTextureObj failed");
    }
}

CUDA_CORE bool CudaCore::TryCreateTextureObj
(
    hipTextureObject_t *obj, 
    const hipResourceDesc *resDesc, 
    const hipTextureDesc *texDesc, 
    const hipResourceViewDesc *resViewDesc
){
    if (CheckCudaErr(hipCreateTextureObject(obj, resDesc, texDesc, resViewDesc)))
    {
#ifndef NDEBUG
        CudaCore::CoutDebug
        ({
            "CUDA create texture object succeeded.", 
            "object:" + std::to_string(*obj),
            "array address:" + ::GetPointerAddress(resDesc->res.array.array),
        });
#endif
        return true;
    }
    else
    {
        hipError_t err = hipGetLastError();
        CudaCore::CoutErr
        ({
            "CUDA create texture object failed.", 
            "code:" + std::to_string(err),
            "reason:" + std::string(hipGetErrorString(err))
        });

        return false;
    }
}

CUDA_CORE void CudaCore::DestroyTextureObj(hipTextureObject_t *obj)
{
    if (CheckCudaErr(hipDestroyTextureObject(*obj)))
    {
#ifndef NDEBUG
        CudaCore::CoutDebug
        ({
            "CUDA destroy texture object succeeded.", 
            "object:" + std::to_string(*obj),
        });
#endif
        *obj = 0;
    }
    else
    {
        hipError_t err = hipGetLastError();
        CudaCore::CoutErr
        ({
            "CUDA destroy texture object failed.", 
            "code:" + std::to_string(err),
            "reason:" + std::string(hipGetErrorString(err))
        });

        throw std::runtime_error("CudaCore::DestroyTextureObj failed");
    }
}

CUDA_CORE bool CudaCore::TryDestroyTextureObj(hipTextureObject_t *obj)
{
    if (CheckCudaErr(hipDestroyTextureObject(*obj)))
    {
#ifndef NDEBUG
        CudaCore::CoutDebug
        ({
            "CUDA destroy texture object succeeded.", 
            "object:" + std::to_string(*obj),
        });
#endif
        *obj = 0;
        return true;
    }
    else
    {
        hipError_t err = hipGetLastError();
        CudaCore::CoutErr
        ({
            "CUDA destroy texture object failed.", 
            "code:" + std::to_string(err),
            "reason:" + std::string(hipGetErrorString(err))
        });

        return false;
    }
}

CUDA_CORE void CudaCore::RegisterResource
(
    hipGraphicsResource_t *resource, ID3D11Resource *d3d11Resource, unsigned int flags
){
    if (CheckCudaErr(cudaGraphicsD3D11RegisterResource(resource, d3d11Resource, flags)))
    {
#ifndef NDEBUG
        CudaCore::CoutDebug
        ({
            "CUDA register resource succeeded.", 
            "resource address:" + ::GetPointerAddress(*resource),
            "ID3D11Resource address:" + ::GetPointerAddress(d3d11Resource),
        });
#endif
    }
    else
    {
        hipError_t err = hipGetLastError();
        CudaCore::CoutErr
        ({
            "CUDA register resource failed.", 
            "code:" + std::to_string(err),
            "reason:" + std::string(hipGetErrorString(err))
        });

        throw std::runtime_error("CudaCore::RegisterResource failed");
    }
}

CUDA_CORE bool CudaCore::TryRegisterResource(hipGraphicsResource_t *resource, ID3D11Resource *d3d11Resource, unsigned int flags)
{
    if (CheckCudaErr(cudaGraphicsD3D11RegisterResource(resource, d3d11Resource, flags)))
    {
#ifndef NDEBUG
        CudaCore::CoutDebug
        ({
            "CUDA register resource succeeded.", 
            "resource address:" + ::GetPointerAddress(*resource),
            "ID3D11Resource address:" + ::GetPointerAddress(d3d11Resource),
        });
#endif
        return true;
    }
    else
    {
        hipError_t err = hipGetLastError();
        CudaCore::CoutErr
        ({
            "CUDA register resource failed.", 
            "code:" + std::to_string(err),
            "reason:" + std::string(hipGetErrorString(err))
        });

        return false;
    }
}

CUDA_CORE void CudaCore::UnregisterResource(hipGraphicsResource_t *resource)
{
    if (CheckCudaErr(hipGraphicsUnregisterResource(*resource)))
    {
#ifndef NDEBUG
        CudaCore::CoutDebug
        ({
            "CUDA unregister resource succeeded.", 
            "resource address:" + ::GetPointerAddress(*resource),
        });
#endif
        *resource = nullptr;
    }
    else
    {
        hipError_t err = hipGetLastError();
        CudaCore::CoutErr
        ({
            "CUDA unregister resource failed.", 
            "code:" + std::to_string(err),
            "reason:" + std::string(hipGetErrorString(err))
        });

        throw std::runtime_error("CudaCore::UnregisterResource failed");
    }
}

CUDA_CORE bool CudaCore::TryUnregisterResource(hipGraphicsResource_t *resource)
{
    if (CheckCudaErr(hipGraphicsUnregisterResource(*resource)))
    {
#ifndef NDEBUG
        CudaCore::CoutDebug
        ({
            "CUDA unregister resource succeeded.", 
            "resource address:" + ::GetPointerAddress(*resource),
        });
#endif
        *resource = nullptr;
        return true;
    }
    else
    {
        hipError_t err = hipGetLastError();
        CudaCore::CoutErr
        ({
            "CUDA unregister resource failed.", 
            "code:" + std::to_string(err),
            "reason:" + std::string(hipGetErrorString(err))
        });

        return false;
    }
}

CUDA_CORE void CudaCore::MapResource(int count, hipGraphicsResource_t *resource, hipStream_t stream)
{
    if (CheckCudaErr(hipGraphicsMapResources(count, resource, stream)))
    {
#ifndef NDEBUG
        CudaCore::CoutDebug
        ({
            "CUDA map resource succeeded.", 
            "resource address:" + ::GetPointerAddress(*resource),
            "stream address:" + ::GetPointerAddress(stream),
            "count:" + std::to_string(count),
        });
#endif
    }
    else
    {
        hipError_t err = hipGetLastError();
        CudaCore::CoutErr
        ({
            "CUDA map resource failed.", 
            "code:" + std::to_string(err),
            "reason:" + std::string(hipGetErrorString(err))
        });

        throw std::runtime_error("CudaCore::MapResource failed");
    }
}

CUDA_CORE bool CudaCore::TryMapResource(int count, hipGraphicsResource_t *resource, hipStream_t stream)
{
    if (CheckCudaErr(hipGraphicsMapResources(count, resource, stream)))
    {
#ifndef NDEBUG
        CudaCore::CoutDebug
        ({
            "CUDA map resource succeeded.", 
            "resource address:" + ::GetPointerAddress(*resource),
            "stream address:" + ::GetPointerAddress(stream),
            "count:" + std::to_string(count),
        });
#endif
        return true;
    }
    else
    {
        hipError_t err = hipGetLastError();
        CudaCore::CoutErr
        ({
            "CUDA map resource failed.", 
            "code:" + std::to_string(err),
            "reason:" + std::string(hipGetErrorString(err))
        });

        return false;
    }
}

CUDA_CORE void CudaCore::UnmapResource(int count, hipGraphicsResource_t *resource, hipStream_t stream)
{
    if (CheckCudaErr(hipGraphicsUnmapResources(count, resource, stream)))
    {
#ifndef NDEBUG
        CudaCore::CoutDebug
        ({
            "CUDA unmap resource succeeded.", 
            "resource address:" + ::GetPointerAddress(*resource),
            "stream address:" + ::GetPointerAddress(stream),
            "count:" + std::to_string(count),
        });
#endif
    }
    else
    {
        hipError_t err = hipGetLastError();
        CudaCore::CoutErr
        ({
            "CUDA unmap resource failed.", 
            "code:" + std::to_string(err),
            "reason:" + std::string(hipGetErrorString(err))
        });

        throw std::runtime_error("CudaCore::UnmapResource failed");
    }
}

CUDA_CORE bool CudaCore::TryUnmapResource(int count, hipGraphicsResource_t *resource, hipStream_t stream)
{
    if (CheckCudaErr(hipGraphicsUnmapResources(count, resource, stream)))
    {
#ifndef NDEBUG
        CudaCore::CoutDebug
        ({
            "CUDA unmap resource succeeded.", 
            "resource address:" + ::GetPointerAddress(*resource),
            "stream address:" + ::GetPointerAddress(stream),
            "count:" + std::to_string(count),
        });
#endif
        return true;
    }
    else
    {
        hipError_t err = hipGetLastError();
        CudaCore::CoutErr
        ({
            "CUDA unmap resource failed.", 
            "code:" + std::to_string(err),
            "reason:" + std::string(hipGetErrorString(err))
        });

        return false;
    }
}

CUDA_CORE void CudaCore::GetMappedPointer(void **devPtr, size_t *size, hipGraphicsResource_t resource)
{
    if (CheckCudaErr(hipGraphicsResourceGetMappedPointer(devPtr, size, resource)))
    {
#ifndef NDEBUG
        CudaCore::CoutDebug
        ({
            "CUDA get mapped pointer succeeded.", 
            "pointer address:" + ::GetPointerAddress(*devPtr),
            "size:" + std::to_string(*size),
            "resource address:" + ::GetPointerAddress(resource),
        });
#endif
    }
    else
    {
        hipError_t err = hipGetLastError();
        CudaCore::CoutErr
        ({
            "CUDA get mapped pointer failed.", 
            "code:" + std::to_string(err),
            "reason:" + std::string(hipGetErrorString(err))
        });

        throw std::runtime_error("CudaCore::GetMappedPointer failed");
    }
}

CUDA_CORE bool CudaCore::TryGetMappedPointer(void **devPtr, size_t *size, hipGraphicsResource_t resource)
{
    if (CheckCudaErr(hipGraphicsResourceGetMappedPointer(devPtr, size, resource)))
    {
#ifndef NDEBUG
        CudaCore::CoutDebug
        ({
            "CUDA get mapped pointer succeeded.", 
            "pointer address:" + ::GetPointerAddress(*devPtr),
            "size:" + std::to_string(*size),
            "resource address:" + ::GetPointerAddress(resource),
        });
#endif
        return true;
    }
    else
    {
        hipError_t err = hipGetLastError();
        CudaCore::CoutErr
        ({
            "CUDA get mapped pointer failed.", 
            "code:" + std::to_string(err),
            "reason:" + std::string(hipGetErrorString(err))
        });

        return false;
    }
}

CUDA_CORE void CudaCore::GetMappedArray
(
    hipArray_t *array, hipGraphicsResource_t resource, unsigned int arrayIndex, unsigned int mipLevel
){
    if (CheckCudaErr(hipGraphicsSubResourceGetMappedArray(array, resource, arrayIndex, mipLevel)))
    {
#ifndef NDEBUG
        CudaCore::CoutDebug
        ({
            "CUDA get mapped array succeeded.", 
            "array address:" + ::GetPointerAddress(*array),
            "resource address:" + ::GetPointerAddress(resource),
            "array index:" + std::to_string(arrayIndex),
            "mip level:" + std::to_string(mipLevel),
        });
#endif
    }
    else
    {
        hipError_t err = hipGetLastError();
        CudaCore::CoutErr
        ({
            "CUDA get mapped array failed.", 
            "code:" + std::to_string(err),
            "reason:" + std::string(hipGetErrorString(err))
        });

        throw std::runtime_error("CudaCore::GetMappedArray failed");
    }
}

CUDA_CORE bool CudaCore::TryGetMappedArray(hipArray_t *array, hipGraphicsResource_t resource, unsigned int arrayIndex, unsigned int mipLevel)
{
    if (CheckCudaErr(hipGraphicsSubResourceGetMappedArray(array, resource, arrayIndex, mipLevel)))
    {
#ifndef NDEBUG
        CudaCore::CoutDebug
        ({
            "CUDA get mapped array succeeded.", 
            "array address:" + ::GetPointerAddress(*array),
            "resource address:" + ::GetPointerAddress(resource),
            "array index:" + std::to_string(arrayIndex),
            "mip level:" + std::to_string(mipLevel),
        });
#endif
        return true;
    }
    else
    {
        hipError_t err = hipGetLastError();
        CudaCore::CoutErr
        ({
            "CUDA get mapped array failed.", 
            "code:" + std::to_string(err),
            "reason:" + std::string(hipGetErrorString(err))
        });

        return false;
    }
}
